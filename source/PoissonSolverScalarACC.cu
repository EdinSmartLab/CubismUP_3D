#include "hip/hip_runtime.h"
//
//  Cubism3D
//  Copyright (c) 2018 CSE-Lab, ETH Zurich, Switzerland.
//  Distributed under the terms of the MIT license.
//
//  Created by Guido Novati (novatig@ethz.ch).
//

//#include "PoissonSolverScalarFFTW_ACC.h"
//#include <hip/hip_runtime_api.h>
#include <array>
#include <vector>
#include <cassert>
using namespace std;

#ifndef _FLOAT_PRECISION_
  #include "accfft_gpu.h"
  typedef accfft_plan_gpu myplan;
  #define accfft_local_size accfft_local_size_dft_r2c_gpu
  #define accfft_plan_dft accfft_plan_dft_3d_r2c_gpu
  #define accfft_delplan accfft_destroy_plan_gpu
  #define accfft_clean accfft_cleanup_gpu
  #define accfft_locsize_dft_r2c accfft_local_size_dft_r2c_gpu
  #define accfft_exec_r2c accfft_execute_r2c_gpu
  #define accfft_exec_c2r accfft_execute_c2r_gpu
  #define MPIREAL MPI_DOUBLE
  typedef double Real;
  typedef double myComplex[2];

#else
  #include "accfft_gpuf.h"
  typedef accfft_plan_gpuf myplan;
  #define accfft_local_size accfft_local_size_dft_r2c_gpuf
  #define accfft_plan_dft accfft_plan_dft_3d_r2c_gpuf
  #define accfft_delplan accfft_destroy_plan_gpu
  #define accfft_clean accfft_cleanup_gpuf
  #define accfft_locsize_dft_r2c accfft_local_size_dft_r2c_gpuf
  #define accfft_exec_r2c accfft_execute_r2c_gpuf
  #define accfft_exec_c2r accfft_execute_c2r_gpuf
  #define MPIREAL MPI_FLOAT
  typedef float Real;
  typedef float myComplex[2];
#endif

#define CUDA_Check(code) do {  \
    if (code != hipSuccess) { \
      printf("DONE DEAD func:%s file:%s:%d %s\n", __func__, \
      __FILE__,__LINE__, hipGetErrorString(code)); \
    } \
} while(0)

__global__
void _fourier_filter_kernel(myComplex*const __restrict__ out,
  const size_t Nx, const size_t Ny, const size_t Nz,
  const size_t nx, const size_t ny, const size_t nz,
  const size_t sx, const size_t sy, const size_t sz,
  const Real wx, const Real wy, const Real wz, const Real fac)
{
  const size_t i = blockDim.x * blockIdx.x + threadIdx.x;
  const size_t j = blockDim.y * blockIdx.y + threadIdx.y;
  const size_t k = blockDim.z * blockIdx.z + threadIdx.z;
  if(i>=nx) return;
  if(j>=ny) return;
  if(k>=nz) return;

  const size_t kx = sx + i, ky = sy + j, kz = sz + k;
  const size_t kkx = kx > Nx/2 ? kx-Nx : kx;
  const size_t kky = ky > Ny/2 ? ky-Ny : ky;
  const size_t kkz = kz > Nz/2 ? kz-Nz : kz;
  const Real rkx = kkx*wx, rky = kky*wy, rkz = kkz*wz;

  const Real kinv =(kkx==0&&kky==0&&kkz==0)? 0 : -fac/(rkx*rkx+rky*rky+rkz*rkz);
  //const Real kinv = -scale*(rkx*rkx+rky*rky+rkz*rkz);
  const size_t index = (i*ny + j)*nz + k;
  out[index][0] *= kinv;
  out[index][1] *= kinv;
}


void _fourier_filter_gpu(myComplex*const __restrict__ out,
  const std::array<size_t,3> N, const std::array<size_t,3> osize,
  const std::array<size_t,3> ostart, const double h)
{
  const std::array<Real,3> wfac = {
    Real(2*M_PI/(h*N[0])),
    Real(2*M_PI/(h*N[1])),
    Real(2*M_PI/(h*N[2]))
  };
  const Real scale = 1./( Real(N[0]) * Real(N[1]) * Real(N[2]) );

  int blocksInX = std::ceil(osize[0] / 4.);
  int blocksInY = std::ceil(osize[1] / 4.);
  int blocksInZ = std::ceil(osize[2] / 4.);

  dim3 Dg(blocksInX, blocksInY, blocksInZ);
  dim3 Db(4, 4, 4);
  _fourier_filter_kernel<<<Dg, Db>>>(
    out, N[0], N[1], N[2],
    osize[0], osize[1], osize[2],
    ostart[0], ostart[1], ostart[2],
    wfac[0], wfac[1], wfac[2], scale);

  hipDeviceSynchronize();
}


__global__ void kPos(const int iSzX, const int iSzY, const int iSzZ,
  const int iStX, const int iStY, const int iStZ, const int nGlobX,
  const int nGlobY, const int nGlobZ, const size_t nZpad, Real*const in_out)
{
  const int i = blockDim.x * blockIdx.x + threadIdx.x;
  const int j = blockDim.y * blockIdx.y + threadIdx.y;
  const int k = blockDim.z * blockIdx.z + threadIdx.z;
  if ( (i >= iSzX) || (j >= iSzY) || (k >= iSzZ) ) return;
  const size_t linidx = k + 2*nZpad*(j + iSzY*i);
  const Real I = i + iStX, J = j + iStY, K = k + iStZ;
  in_out[linidx] = K + nGlobZ * (J + nGlobY * I);
}

__global__ void kGreen(const int iSzX, const int iSzY, const int iSzZ,
  const int iStX, const int iStY, const int iStZ,
  const int nGlobX, const int nGlobY, const int nGlobZ, const size_t nZpad,
  const Real fac, const Real h, Real*const in_out)
{
  const int i = blockDim.x * blockIdx.x + threadIdx.x;
  const int j = blockDim.y * blockIdx.y + threadIdx.y;
  const int k = blockDim.z * blockIdx.z + threadIdx.z;
  if ( (i >= iSzX) || (j >= iSzY) || (k >= iSzZ) ) return;
  const size_t linidx = k + 2*nZpad*(j + iSzY*i);
  const int I = i + iStX, J = j + iStY, K = k + iStZ;
  const Real xi = I>=nGlobX? 2*nGlobX-1 - I : I;
  const Real yi = J>=nGlobY? 2*nGlobY-1 - J : J;
  const Real zi = K>=nGlobZ? 2*nGlobZ-1 - K : K;
  const Real r = std::sqrt(xi*xi + yi*yi + zi*zi);
  if(r > 0) in_out[linidx] = fac / r;
  // G = r_eq^2 / 2 = std::pow(3/8/pi/sqrt(2))^(2/3) * h^2
  else      in_out[linidx] = Real(0.1924173658) * h * h;
  //else      in_out[linidx] = fac;
}

__global__ void kCopyC2R(const int oSzX,const int oSzY,const int oSzZ,
  const Real norm, const size_t nZpad, const myComplex*const G_hat, Real*const m_kernel)
{
  const int i = threadIdx.x + blockIdx.x * blockDim.x;
  const int j = threadIdx.y + blockIdx.y * blockDim.y;
  const int k = threadIdx.z + blockIdx.z * blockDim.z;
  if ( (i >= oSzX) || (j >= oSzY) || (k >= oSzZ) ) return;
  const size_t linidx = (i*oSzY + j)*nZpad + k;
  m_kernel[linidx] = G_hat[linidx][0] * norm;
}

__global__ void kFreespace(const int oSzX, const int oSzY, const int oSzZ,
  const size_t nZpad, const Real*const G_hat, myComplex*const in_out)
{
  const int i = threadIdx.x + blockIdx.x * blockDim.x;
  const int j = threadIdx.y + blockIdx.y * blockDim.y;
  const int k = threadIdx.z + blockIdx.z * blockDim.z;
  if ( (i >= oSzX) || (j >= oSzY) || (k >= oSzZ) ) return;
  const size_t linidx = (i*oSzY + j)*nZpad + k;
  in_out[linidx][0] *= G_hat[linidx];
  in_out[linidx][1] *= G_hat[linidx];
  //in_out[linidx][0] *= 1./(2047 * 511 * 255);
  //in_out[linidx][1] *= 1./(2047 * 511 * 255);
}

static inline int getSize(MPI_Comm comm) {
  int ret; MPI_Comm_size(comm, &ret); return ret;
}
static inline int getRank(MPI_Comm comm) {
  int ret; MPI_Comm_rank(comm, &ret); return ret;
}

void dSolveFreespace(void*const P, const int nx,const int ny,const int nz,
  const int locx,const int locy,const int locz, const MPI_Comm comm,
  const int ox,const int oy,const int oz, const Real*const G_hat,
  Real*const cub_rhs, Real*const fft_rhs, Real*const gpu_rhs)
{
  const int mx = 2*nx -1, my = 2*ny -1, mz = 2*nz -1, mz_pad = mz/2 +1;
  const int mpisize = getSize(comm), mpirank = getRank(comm);

  int pos[3], dst[3];
  MPI_Cart_coords(comm, mpirank, 3, pos);
  int szFft[3] = { (mx+1)/mpisize, ny, nz }, start[3]={0,0,0};
  int szCup[3] = { std::min(szFft[0], locx), locy, locz };

  MPI_Datatype submat;
  MPI_Type_create_subarray(3, szFft, szCup, start, MPI_ORDER_C,MPIREAL,&submat);
  MPI_Type_commit(&submat);
  // MPI transfer of data from CUP distribution to 1D-padded FFT distribution
  {
    memset(fft_rhs, 0, szFft[0]*szFft[1]*szFft[2] * sizeof(Real) );
    vector<MPI_Request> reqs = vector<MPI_Request>(mpisize*2, MPI_REQUEST_NULL);
    const int m_ind =  pos[0]   * locx, m_pos =  mpirank   * szFft[0];
    const int m_nxt = (pos[0]+1)* locx, m_end = (mpirank+1)* szFft[0];
    for(int i=0; i<mpisize; i++)
    {
      MPI_Cart_coords(comm, i, 3, dst); // assert(dst[1]==0 && dst[2]==0);
      const int i_ind =  dst[0]   * locx, i_pos =  i   * szFft[0];
      const int i_nxt = (dst[0]+1)* locx, i_end = (i+1)* szFft[0];
      // test if rank needs to send to i its rhs:
      if( i_pos < m_nxt && m_ind < i_end )
      {
        const int tag = i + mpirank * mpisize;
        const size_t shiftx = std::max(i_pos - m_ind, 0);
        const size_t ptr = szCup[2] * szCup[1] * shiftx;
        MPI_Isend(cub_rhs + ptr, 1, submat, i, tag, comm, &reqs[2*i]);
      }
      // test if rank needs to recv to i's rhs:
      if( m_pos < i_nxt && i_ind < m_end )
      {
        const int tag = mpirank + i * mpisize;
        const size_t shiftx = std::max(i_ind - m_pos, 0);
        const size_t ptr = dst[2]*szCup[2] +nz*(dst[1]*szCup[1] +ny*shiftx);
        MPI_Irecv(fft_rhs + ptr, 1, submat, i, tag, comm, &reqs[2*i + 1]);
      }
    }
    MPI_Waitall(mpisize*2, reqs.data(), MPI_STATUSES_IGNORE);
  }

  // ranks that do not contain only zero-padding, transfer RHS to GPU
  if(mpirank < mpisize/2)
  {
    #if 1
      hipMemcpy3DParms p = {};
      p.srcPos.x=0; p.srcPos.y=0; p.srcPos.z=0; p.dstPos.x=0; p.dstPos.y=0; p.dstPos.z=0;
      p.dstPtr = make_hipPitchedPtr(gpu_rhs, 2*mz_pad*sizeof(Real), 2*mz_pad, my);
      p.srcPtr = make_hipPitchedPtr(fft_rhs, szFft[2]*sizeof(Real), szFft[2], szFft[1]);
      p.extent = make_hipExtent(szFft[2]*sizeof(Real), szFft[1], szFft[0]);
      p.kind = hipMemcpyHostToDevice;
      CUDA_Check(hipMemcpy3D(&p));
    #else
      for(int i=0; i<szFft[0]; i++) {
        CUDA_Check(hipMemcpy2D(
          gpu_rhs + 2*mz_pad*my*i, 2*mz_pad*sizeof(Real),
          fft_rhs + szFft[2]*szFft[1]*i, szFft[2]*sizeof(Real),
          szFft[2]*sizeof(Real), szFft[1], // sizes
          hipMemcpyHostToDevice) );
      }
    #endif
    //CUDA_Check(hipDeviceSynchronize());
  }

  // solve Poisson in padded space
  {
    accfft_exec_r2c(static_cast<myplan*>(P), gpu_rhs, (myComplex*) gpu_rhs);
    //CUDA_Check(hipDeviceSynchronize());
    dim3 dB(4, 4, 4);
    dim3 dG(std::ceil(ox/4.), std::ceil(oy/4.), std::ceil(oz/4.));
    kFreespace <<<dG, dB>>> (ox,oy,oz, mz_pad, G_hat, (myComplex*) gpu_rhs);
    //CUDA_Check(hipDeviceSynchronize());
    accfft_exec_c2r(static_cast<myplan*>(P), (myComplex*) gpu_rhs, gpu_rhs);
    //CUDA_Check(hipDeviceSynchronize());
  }

  // ranks that do not contain extended solution, transfer SOL to CPU
  if(mpirank < mpisize/2)
  {
    #if 1
      hipMemcpy3DParms p = {};
      p.srcPos.x=0; p.srcPos.y=0; p.srcPos.z=0; p.dstPos.x=0; p.dstPos.y=0; p.dstPos.z=0;
      p.srcPtr = make_hipPitchedPtr(gpu_rhs, 2*mz_pad*sizeof(Real), 2*mz_pad, my);
      p.dstPtr = make_hipPitchedPtr(fft_rhs, szFft[2]*sizeof(Real), szFft[2], szFft[1]);
      p.extent = make_hipExtent(szFft[2]*sizeof(Real), szFft[1], szFft[0]);
      p.kind = hipMemcpyDeviceToHost;
      CUDA_Check(hipMemcpy3D(&p));
    #else
      for(int i=0; i<szFft[0]; i++) {
        CUDA_Check(hipMemcpy2D(
          fft_rhs + szFft[2]*szFft[1]*i, szFft[2]*sizeof(Real),
          gpu_rhs + 2*mz_pad*my*i, 2*mz_pad*sizeof(Real),
          szFft[2]*sizeof(Real), szFft[1], // sizes
          hipMemcpyDeviceToHost) );
      }
    #endif
    //CUDA_Check(hipDeviceSynchronize());
  }
  {
    //for(size_t i=0; i<szFft[0]; i++)
    //for(size_t j=0; j<szFft[1]; j++)
    //for(size_t k=0; k<szFft[2]; k++)
    //  fft_rhs[k+szFft[2]*(j+szFft[1]*i)] = k+mz*(j+my*(i+mpirank*szFft[0]));
  }
  // MPI transfer of data from CUP distribution to 1D-padded FFT distribution
  {
    vector<MPI_Request> reqs = vector<MPI_Request>(mpisize*2, MPI_REQUEST_NULL);
    const int m_ind =  pos[0]   * locx, m_pos =  mpirank   * szFft[0];
    const int m_nxt = (pos[0]+1)* locx, m_end = (mpirank+1)* szFft[0];
    for(int i=0; i<mpisize; i++)
    {
      MPI_Cart_coords(comm, i, 3, dst);
      const int i_ind =  dst[0]   * locx, i_pos =  i   * szFft[0];
      const int i_nxt = (dst[0]+1)* locx, i_end = (i+1)* szFft[0];
      // test if rank needs to send to i its rhs:
      if( i_pos < m_nxt && m_ind < i_end )
      {
        const int tag = i + mpirank * mpisize;
        const size_t shiftx = std::max(i_pos - m_ind, 0);
        const size_t ptr = szCup[2] * szCup[1] * shiftx;
        MPI_Irecv(cub_rhs + ptr, 1, submat, i, tag, comm, &reqs[2*i]);
      }
      // test if rank needs to recv to i's rhs:
      if( m_pos < i_nxt && i_ind < m_end )
      {
        const int tag = mpirank + i * mpisize;
        const size_t shiftx = std::max(i_ind - m_pos, 0);
        const size_t ptr = dst[2]*szCup[2] +nz*(dst[1]*szCup[1] +ny*shiftx);
        MPI_Isend(fft_rhs + ptr, 1, submat, i, tag, comm, &reqs[2*i + 1]);
      }
    }
    MPI_Waitall(mpisize*2, reqs.data(), MPI_STATUSES_IGNORE);
  }
  MPI_Type_free(&submat);
}

void initGreen(const int *isz,const int *osz,const int *ist,const int *ost,
  const int nx,const int ny,const int nz, const Real h, myplan* const fwd,
  Real*const m_kernel, Real*const gpu_rhs)
{
  const int mx = 2*nx -1, my = 2*ny -1, mz = 2*nz -1, mz_pad = mz/2 +1;
  {
    const Real fac = - h * h / ( 4.0 * M_PI );
    dim3 dB(4, 4, 4);
    dim3 dG(std::ceil(isz[0]/4.), std::ceil(isz[1]/4.), std::ceil(isz[2]/4.));
    //cout<<isz[0]<<" "<<isz[1]<<" "<<isz[2]<<" "<< ist[0]<<" "<<ist[1]<<" "<<ist[2]<<" "<<nx<<" "<<ny<<" "<<nz<<" "<<mz_pad<<endl;
    kGreen<<<dG, dB>>> (isz[0],isz[1],isz[2], ist[0],ist[1],ist[2],
      nx, ny, nz, mz_pad, fac, h, gpu_rhs);
    CUDA_Check(hipDeviceSynchronize());
  }

  accfft_exec_r2c(fwd, gpu_rhs, (myComplex*) gpu_rhs);
  CUDA_Check(hipDeviceSynchronize());

  {
    const Real norm = 1.0 / ((Real) mx * (Real) my * (Real) mz);
    dim3 dB(4, 4, 4);
    dim3 dG(std::ceil(osz[0]/4.), std::ceil(osz[1]/4.), std::ceil(osz[2]/4.));
    kCopyC2R<<<dG, dB>>> (osz[0],osz[1],osz[2], norm, mz_pad,
      (myComplex*)gpu_rhs, m_kernel);
    CUDA_Check(hipDeviceSynchronize());
  }
  //{
  //  dim3 dB(4, 4, 4);
  //  dim3 dG(std::ceil(isz[0]/4.), std::ceil(isz[1]/4.), std::ceil(isz[2]/4.));
  //  kPos<<<dG, dB>>> (isz[0],isz[1],isz[2], ist[0],ist[1],ist[2], mx,my,mz, mz_pad, gpu_rhs);
  //}
}

MPI_Comm my_accfft_create_comm(MPI_Comm C, int c_dims[2]) {
  MPI_Comm ret;
  accfft_create_comm( C, c_dims, &ret);
  return ret;
}

void my_cudaFree(Real* const ptr) {  hipFree(ptr); }
void my_accfft_cleanup(void* const plan) {
  myplan * inp = static_cast<myplan*>(plan);
  accfft_destroy_plan_gpu(inp);
  accfft_clean();
}
