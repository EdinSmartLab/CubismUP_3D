#include "hip/hip_runtime.h"
//
//  Cubism3D
//  Copyright (c) 2018 CSE-Lab, ETH Zurich, Switzerland.
//  Distributed under the terms of the MIT license.
//
//  Created by Guido Novati (novatig@ethz.ch).
//

//#include "PoissonSolverScalarFFTW_ACC.h"
//#include <hip/hip_runtime_api.h>
#include <array>
#include <vector>
using namespace std;

#ifndef _FLOAT_PRECISION_
  #include "accfft_gpu.h"
  typedef accfft_plan_gpu myplan;
  #define accfft_local_size accfft_local_size_dft_r2c_gpu
  #define accfft_plan_dft accfft_plan_dft_3d_r2c_gpu
  #define accfft_delplan accfft_destroy_plan_gpu
  #define accfft_clean accfft_cleanup_gpu
  #define accfft_locsize_dft_r2c accfft_local_size_dft_r2c_gpu
  #define accfft_exec_r2c accfft_execute_r2c_gpu
  #define accfft_exec_c2r accfft_execute_c2r_gpu
  #define MPIREAL MPI_DOUBLE
  typedef double Real;
  typedef double myComplex[2];

#else
  #include "accfft_gpuf.h"
  typedef accfft_plan_gpuf myplan;
  #define accfft_local_size accfft_local_size_dft_r2c_gpuf
  #define accfft_plan_dft accfft_plan_dft_3d_r2c_gpuf
  #define accfft_delplan accfft_destroy_plan_gpu
  #define accfft_clean accfft_cleanup_gpuf
  #define accfft_locsize_dft_r2c accfft_local_size_dft_r2c_gpuf
  #define accfft_exec_r2c accfft_execute_r2c_gpuf
  #define accfft_exec_c2r accfft_execute_c2r_gpuf
  #define MPIREAL MPI_FLOAT
  typedef float Real;
  typedef float myComplex[2];
#endif

__global__
void _fourier_filter_kernel(myComplex*const __restrict__ out,
  const size_t Nx, const size_t Ny, const size_t Nz,
  const size_t nx, const size_t ny, const size_t nz,
  const size_t sx, const size_t sy, const size_t sz,
  const Real wx, const Real wy, const Real wz, const Real fac)
{
  const size_t i = blockDim.x * blockIdx.x + threadIdx.x;
  const size_t j = blockDim.y * blockIdx.y + threadIdx.y;
  const size_t k = blockDim.z * blockIdx.z + threadIdx.z;
  if(i>=nx) return;
  if(j>=ny) return;
  if(k>=nz) return;

  const size_t kx = sx + i, ky = sy + j, kz = sz + k;
  const size_t kkx = kx > Nx/2 ? kx-Nx : kx;
  const size_t kky = ky > Ny/2 ? ky-Ny : ky;
  const size_t kkz = kz > Nz/2 ? kz-Nz : kz;
  const Real rkx = kkx*wx, rky = kky*wy, rkz = kkz*wz;

  const Real kinv =(kkx==0&&kky==0&&kkz==0)? 0 : -fac/(rkx*rkx+rky*rky+rkz*rkz);
  //const Real kinv = -scale*(rkx*rkx+rky*rky+rkz*rkz);
  const size_t index = (i*ny + j)*nz + k;
  out[index][0] *= kinv;
  out[index][1] *= kinv;
}


void _fourier_filter_gpu(myComplex*const __restrict__ out,
  const std::array<size_t,3> N, const std::array<size_t,3> osize,
  const std::array<size_t,3> ostart, const double h)
{
  const std::array<Real,3> wfac = {
    Real(2*M_PI/(h*N[0])),
    Real(2*M_PI/(h*N[1])),
    Real(2*M_PI/(h*N[2]))
  };
  const Real scale = 1./Real(N[0]*N[1]*N[2]);

  int blocksInX = std::ceil(osize[0] / 4.);
  int blocksInY = std::ceil(osize[1] / 4.);
  int blocksInZ = std::ceil(osize[2] / 4.);

  dim3 Dg(blocksInX, blocksInY, blocksInZ);
  dim3 Db(4, 4, 4);
  _fourier_filter_kernel<<<Dg, Db>>>(
    out, N[0], N[1], N[2],
    osize[0], osize[1], osize[2],
    ostart[0], ostart[1], ostart[2],
    wfac[0], wfac[1], wfac[2], scale);

  hipDeviceSynchronize();
}


__global__ void kGreen(const int iSzX, const int iSzY, const int iSzZ,
  const int iStX, const int iStY, const int iStZ,
  const int nGlobX, const int nGlobY, const int nGlobZ, const int nZpad,
  const Real fac, const Real h, Real*const in_out) {
  unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
  unsigned int j = blockDim.y * blockIdx.y + threadIdx.y;
  unsigned int k = blockDim.z * blockIdx.z + threadIdx.z;
  if ( (i >= iSzX) || (j >= iSzY) || (k >= iSzZ) ) return;
  const size_t linidx = k + nZpad*(j + iSzY*i);
  const int I = i + iStX, J = j + iStY, K = k + iStZ;
  const Real xi = I>=nGlobX? 2*nGlobX-1 - I : I;
  const Real yi = J>=nGlobY? 2*nGlobY-1 - J : J;
  const Real zi = K>=nGlobZ? 2*nGlobZ-1 - k : k;
  const Real r = std::sqrt(xi*xi + yi*yi + zi*zi);
  if(r > 0) in_out[linidx] = fac / r;
  // G = r_eq^2 / 2 = std::pow(3/8/pi/sqrt(2))^(2/3) * h^2
  else      in_out[linidx] = 0.1924173658 * h * h;
}

__global__ void kCopyC2R(const int oSzX, const int oSzY, const int oSzZ,
  const Real norm, const myComplex*const G_hat, Real*const m_kernel) {
  const int i = threadIdx.x + blockIdx.x * blockDim.x;
  const int j = threadIdx.y + blockIdx.y * blockDim.y;
  const int k = threadIdx.z + blockIdx.z * blockDim.z;
  if ( (i >= oSzX) || (j >= oSzY) || (k >= oSzZ) ) return;
  const int linidx = (i*oSzY + j)*oSzZ + k;
  m_kernel[linidx] = G_hat[linidx][0] * norm;
}

__global__ void kFreespace(const int oSzX, const int oSzY, const int oSzZ,
  const Real*const G_hat, myComplex*const in_out) {
  const int i = threadIdx.x + blockIdx.x * blockDim.x;
  const int j = threadIdx.y + blockIdx.y * blockDim.y;
  const int k = threadIdx.z + blockIdx.z * blockDim.z;
  if ( (i >= oSzX) || (j >= oSzY) || (k >= oSzZ) ) return;
  const int linidx = (i*oSzY + j)*oSzZ + k;
  in_out[linidx][0] *= G_hat[linidx]; in_out[linidx][1] *= G_hat[linidx];
}

void initGreen(const int nx, const int ny, const int nz,
  const Real h, Real*const m_kernel, const MPI_Comm comm)
{
  const int mx = 2*nx -1, my = 2*ny -1, mz = 2*nz -1, mz_pad = (mz/2 +1)*2;
  int M[3] = {mx, my, mz};

  int iSz[3], oSz[3], iSt[3], oSt[3];
  const size_t allocM = accfft_locsize_dft_r2c(M, iSz,iSt,oSz,oSt, comm);
  Real* tmp = nullptr;
  hipMalloc((void**) &tmp, allocM);

  {
    const Real fac = - h * h / ( 4*M_PI );
    dim3 dB(4, 4, 4);
    dim3 dG(std::ceil(iSz[0]/4.), std::ceil(iSz[1]/4.), std::ceil(iSz[2]/4.));
    kGreen<<<dG, dB>>> (iSz[0],iSz[1],iSz[2], iSt[0],iSt[1],iSt[2],
      nx, ny, nz, mz_pad, fac, h, tmp);
  }
  {
    myplan* fwd = accfft_plan_dft(M, tmp, tmp, comm, ACCFFT_MEASURE);
    accfft_exec_r2c(fwd, tmp, (myComplex*) tmp);
    accfft_delplan(fwd);
  }
  {
    const Real norm = 1.0 / (mx * my * mz);
    dim3 dB(4, 4, 4);
    dim3 dG(std::ceil(oSz[0]/4.), std::ceil(oSz[1]/4.), std::ceil(oSz[2]/4.));
    kCopyC2R<<<dG, dB>>> (iSz[0],iSz[1],iSz[2], norm, (myComplex*)tmp, m_kernel);
  }
  hipFree(tmp);
}

static inline int getSize(MPI_Comm comm) {
  int ret; MPI_Comm_size(comm, &ret); return ret;
}
static inline int getRank(MPI_Comm comm) {
  int ret; MPI_Comm_rank(comm, &ret); return ret;
}

void dSolveFreespace(void*const P, const int nx,const int ny,const int nz,
  const int locx,const int locy,const int locz, const MPI_Comm comm,
  const int ox,const int oy,const int oz, const Real*const G_hat,
  Real*const cub_rhs, Real*const fft_rhs, Real*const gpu_rhs, myComplex*const phi)
{
  const int mx = 2*nx -1, my = 2*ny -1, mz = 2*nz -1, mz_pad = (mz/2 +1)*2;
  const int mpisize = getSize(comm), mpirank = getRank(comm);

  int pos[3], dst[3];
  MPI_Cart_coords(comm, mpirank, 3, pos);
  int szFft[3] = { (mx+1)/mpisize, ny, nz }, start[3]={0,0,0};
  int szCup[3] = { std::min(szFft[0], locx), locy, locz };

  MPI_Datatype submat;
  MPI_Type_create_subarray(3, szFft, szCup, start, MPI_ORDER_C,MPIREAL,&submat);
  MPI_Type_commit(&submat);

  {
    vector<MPI_Request> reqs = vector<MPI_Request>(mpisize*2, MPI_REQUEST_NULL);
    const int m_ind =  pos[0]   * locx, m_pos =  mpirank   * szFft[0];
    const int m_nxt = (pos[0]+1)* locx, m_end = (mpirank+1)* szFft[0];
    for(int i=0; i<mpisize; i++)
    {
      MPI_Cart_coords(comm, i, 3, dst);
      const int i_ind =  dst[0]   * locx, i_pos =  i   * szFft[0];
      const int i_nxt = (dst[0]+1)* locx, i_end = (i+1)* szFft[0];
      // test if rank needs to send to i its rhs:
      if( i_pos < m_nxt && m_ind < i_end ) {
        const int tag = i + mpirank * mpisize;
        const int shiftx = std::max(i_pos - m_ind, 0);
        const int ptr = szCup[2] * szCup[1] * shiftx;
        MPI_Isend(cub_rhs + ptr, 1, submat, i, tag, comm, &reqs[2*i]);
      }
      // test if rank needs to recv to i's rhs:
      if( m_pos < i_nxt && i_ind < m_end ) {
        const int tag = mpirank + i * mpisize;
        const int shiftx = std::max(i_ind - m_pos, 0);
        const int ptr = dst[2]*szCup[2] +nz*(dst[1]*szCup[1] +ny*shiftx);
        MPI_Irecv(fft_rhs + ptr, 1, submat, i, tag, comm, &reqs[2*i + 1]);
      }
    }
    MPI_Waitall(mpisize*2, reqs.data(), MPI_STATUSES_IGNORE);
  }
  if(mpirank < mpisize/2) {
    hipMemcpy3DParms p = { 0 };
    p.srcPos.x=0; p.srcPos.y=0; p.srcPos.z=0;
    p.dstPos.x=0; p.dstPos.y=0; p.dstPos.z=0;
    p.srcPtr.ptr   = fft_rhs;  p.srcPtr.pitch = szFft[2] * sizeof(Real);
    p.srcPtr.xsize = szFft[2]; p.srcPtr.ysize = szFft[1];
    p.dstPtr.ptr  = gpu_rhs; p.dstPtr.pitch = oz * sizeof(Real);
    p.dstPtr.xsize = oz; p.dstPtr.ysize = oy;
    p.kind = hipMemcpyHostToDevice; p.extent.width = szFft[2] * sizeof(Real);
    p.extent.height = szFft[1]; p.extent.depth = szFft[0];
    hipMemcpy3D(&p);
  }
  {
    accfft_exec_r2c(static_cast<myplan*>(P), gpu_rhs, phi);
    dim3 dB(4, 4, 4);
    dim3 dG(std::ceil(ox/4.), std::ceil(oy/4.), std::ceil(oz/4.));
    kFreespace <<<dG, dB>>> (ox,oy,oz, G_hat, phi);
    accfft_exec_c2r(static_cast<myplan*>(P), phi, gpu_rhs);
  }
  if(mpirank < mpisize/2) {
    hipMemcpy3DParms p = { 0 };
    p.srcPos.x=0; p.srcPos.y=0; p.srcPos.z=0;
    p.dstPos.x=0; p.dstPos.y=0; p.dstPos.z=0;
    p.dstPtr.ptr   = fft_rhs;  p.dstPtr.pitch = szFft[2] * sizeof(Real);
    p.dstPtr.xsize = szFft[2]; p.dstPtr.ysize = szFft[1];
    p.srcPtr.ptr  = gpu_rhs; p.srcPtr.pitch = oz * sizeof(Real);
    p.srcPtr.xsize = oz; p.srcPtr.ysize = oy;
    p.kind = hipMemcpyDeviceToHost; p.extent.width = szFft[2] * sizeof(Real);
    p.extent.height = szFft[1]; p.extent.depth = szFft[0];
    hipMemcpy3D(&p);
  }
  {
    vector<MPI_Request> reqs = vector<MPI_Request>(mpisize*2, MPI_REQUEST_NULL);
    const int m_ind =  pos[0]   * locx, m_pos =  mpirank   * szFft[0];
    const int m_nxt = (pos[0]+1)* locx, m_end = (mpirank+1)* szFft[0];
    for(int i=0; i<mpisize; i++)
    {
      MPI_Cart_coords(comm, i, 3, dst);
      const int i_ind =  dst[0]   * locx, i_pos =  i   * szFft[0];
      const int i_nxt = (dst[0]+1)* locx, i_end = (i+1)* szFft[0];
      // test if rank needs to send to i its rhs:
      if( i_pos < m_nxt && m_ind < i_end ) {
        const int tag = i + mpirank * mpisize;
        const int shiftx = std::max(i_pos - m_ind, 0);
        const int ptr = szCup[2] * szCup[1] * shiftx;
        MPI_Irecv(cub_rhs + ptr, 1, submat, i, tag, comm, &reqs[2*i]);
      }
      // test if rank needs to recv to i's rhs:
      if( m_pos < i_nxt && i_ind < m_end ) {
        const int tag = mpirank + i * mpisize;
        const int shiftx = std::max(i_ind - m_pos, 0);
        const int ptr = dst[2]*szCup[2] +nz*(dst[1]*szCup[1] +ny*shiftx);
        MPI_Isend(fft_rhs + ptr, 1, submat, i, tag, comm, &reqs[2*i + 1]);
      }
    }
    MPI_Waitall(mpisize*2, reqs.data(), MPI_STATUSES_IGNORE);
  }
  MPI_Type_free(&submat);
}

MPI_Comm my_accfft_create_comm(MPI_Comm C, int c_dims[2]) {
  MPI_Comm ret;
  accfft_create_comm( C, c_dims, &ret);
  return ret;
}

Real* my_cudaMalloc(const size_t size) {
  Real* ret;
  hipMalloc((void**) &ret, size);
  return ret;
}
void my_cudaFree(Real* const ptr) {  hipFree(ptr); }
void my_accfft_cleanup(void* const plan) {
  myplan * inp = static_cast<myplan*>(plan);
  accfft_destroy_plan_gpu(inp);
  accfft_clean();
}

size_t my_accfft_local_size(int M[3], int isz[3], int ist[3], int osz[3],
  int ost[3], MPI_Comm c_comm) {
    return accfft_local_size(M, isz, ist, osz, ost, c_comm);
}

void* my_accfft_plan_dft(int M[3], Real* gpurhs, MPI_Comm c_comm) {
  myplan * ret = accfft_plan_dft(M, gpurhs, gpurhs, c_comm, ACCFFT_MEASURE);
  return static_cast<void*>(ret);
}

void my_cudaMemset_zero(Real* const ptr, const size_t size) {
  hipMemset(ptr, 0, size );
}
